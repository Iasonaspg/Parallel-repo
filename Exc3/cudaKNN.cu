#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math.h>


#define CHECK(call) \
{                    \
    const hipError_t error = call; \
    if (error != hipSuccess){       \
        printf("Error: %s:%d, ", __FILE__, __LINE__);  \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
} \

// Kernel to change 1 with 0 because we want values inside [0,1) and currand produces (0,1]
__global__ void corr(float* ptr, size_t N){

    int index = threadIdx.x + blockDim.x*blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i=index; i<N; i+=stride){
        if (ptr[i] == 1){
            ptr[i] = 0;
        }
    }
}

// Kernel to print points from a big array
__global__ void check(float* ptr, size_t N, int len){

    int index = threadIdx.x + blockDim.x*blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i=index; i<len; i+=stride){
        printf("Point[%d]: (%f,%f,%f)\n",i,ptr[i],ptr[i+N],ptr[i+2*N]);
    }

}

// Kernel to initialize a big array with a specific number. I eventually didn't use it
__global__ void init(float* ptr, int len, int num){

    int index = threadIdx.x + blockDim.x*blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i=index; i<len; i+=stride){
        ptr[i] = num;
    }
}

// Kernel to find the nearest neighbor of a point
__global__ void findNN(float* queries, int len, float* boxesC, int len1, int elements, float side, int* aux, int d, float* distances){

    int index = threadIdx.x + blockDim.x*blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i=index; i<len; i+=stride){
        int coX,coY,coZ;
        int coXtmp,coYtmp,coZtmp;
        int start,boxMin;
        float distance, sum;
        coX = queries[i]/side;
        coY = queries[i+len]/side;
        coZ = queries[i+2*len]/side;
        int box = coX + coY*d + coZ*d*d;    // calculate the box in which the point belongs to
        // printf("Box inside kernel: %d\n",box);
        // printf("Queries inside kernel: (%f,%f,%f)\n",queries[i],queries[i+len],queries[i+2*len]);
        float min = 10000000;
        int pos = -1;
        boxMin = -1;
        start = box*elements;   // Starting point inside the big 1-d array for the desired box
        for (int j=0; j<aux[box];j++){
            sum = 0;
            for (int k=0; k<3; k++){
                sum += powf(queries[i+k*len]-boxesC[start+j+k*len1],2);
            }
            distance = powf(sum,0.5);
            //printf("Distance: %f\n", distance);
            if (distance < min){
                min = distance;     // Calculate the min distance inside the box
                pos = j;
                boxMin = box;
            }
        }

        // Check if I have to search adjacent boxes
        if ((fabsf(queries[i+2*len]-(coZ+1)*side)<min) || (fabsf(queries[i+2*len]-coZ*side)<min) || (fabsf(queries[i+len]-(coY+1)*side)<min) || (fabsf(queries[i+len]-coY*side)<min) || (fabsf(queries[i]-(coX+1)*side)<min) || (fabsf(queries[i]-coX*side)<min)){
            for (int dx=-1;dx<=1;dx++){
                coXtmp = coX + dx;
                if (coXtmp>=d || coXtmp<0){
                    continue;
                }
                for (int dy=-1;dy<=1;dy++){
                    coYtmp = coY + dy;
                    if (coYtmp>=d || coYtmp<0){
                        continue;
                    }
                    for (int dz=-1;dz<=1;dz++){
                        coZtmp = coZ + dz;
                        if (coZtmp>=d || coZtmp<0){
                            continue;
                        }
                        if (dz==0 && dx==0 && dy==0){
                            continue;
                        }
                        box = coXtmp + coYtmp*d + coZtmp*d*d;
                        // printf("Box number: %d\n",box);
                        start = box*elements;
                        for (int j=0; j<aux[box];j++){
                            sum = 0;
                            for (int k=0; k<3; k++){
                                sum += powf(queries[i+k*len]-boxesC[start+j+k*len1],2);
                            }
                            distance = powf(sum,0.5);
                            //printf("Distance: %f\n", distance);
                            if (distance < min){
                                min = distance;
                                pos = j;
                                boxMin = box;
                            }
                        }
                    }
                }
            }
        } 
        // printf("Min distance from (%f,%f,%f): %f at box: %d and pos: %d\n",queries[i],queries[i+len],queries[i+2*len],min,boxMin,pos);
        int start1 = boxMin*elements;
        // printf("Nearest neighbor of (%f,%f,%f): (%f,%f,%f)\n",queries[i],queries[i+len],queries[i+2*len],boxesC[start1+pos],boxesC[start1+pos+len1],boxesC[start1+pos+2*len1]);
        distances[i] = min;  // Store the final min distance
    }
}

// Kernel that calculates minimum distance in a brute-force way
__global__ void validate(float* queries, float* corpus, int len, int* flag, float* distances){

    int index = threadIdx.x + blockDim.x*blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int j=index;j<len;j+=stride){
        float min = 10000;
        for (int i=0;i<len;i++){
            float sum = 0;
            for (int k=0; k<3; k++){
                sum += powf(queries[j+k*len]-corpus[i+k*len],2);
            }
            float distance = powf(sum,0.5);
            if (distance < min){
                min = distance;
            }
        }
        // printf("Minimum distance for query[%d]: %f\n",j,min);
        if (min != distances[j]) {
            *flag = 1;
        }
    }

}

// Function that returns current time in seconds
double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}



int main(int argc, char** argv){

    size_t Nq = (1<<atoi(argv[1]));     // Nq=Nc
    int d = (1<<atoi(argv[2]));
    float side = 1/((float) d);
    printf("Side length of each box: %f\n",side);
    
    
    hiprandGenerator_t gen;
    float *queries, *corpus;

    // Generate Uniform Points
    CHECK(hipMallocManaged(&queries, 3*Nq*sizeof(float)));
    CHECK(hipMallocManaged(&corpus, 3*Nq*sizeof(float)));
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen,time(NULL));
    
    double start = cpuSecond();
    hiprandGenerateUniform(gen, queries, 3*Nq);
    hiprandGenerateUniform(gen, corpus, 3*Nq);
    corr<<<((Nq+1023)/1024),1024>>>(queries,3*Nq);
    corr<<<((Nq+1023)/1024),1024>>>(corpus,3*Nq);
    CHECK(hipDeviceSynchronize());
    double elaps = cpuSecond() - start;
    printf("Time elapsed for generating points: %f seconds\n",elaps);
    //check<<<1,32>>>(queries,Nq,Nq);
    //check<<<1,32>>>(corpus,Nq,Nq);


    int newNq = 8*Nq;
    float* boxesC;
    CHECK(hipMallocManaged(&boxesC, 3*newNq*sizeof(float)));   // Array that is going to store the points
    
    int* auxC;                                                  // Array that will store the number of points inside each box
    CHECK(hipMallocManaged(&auxC, d*d*d*sizeof(int)));
    CHECK(hipMemset(auxC,0,d*d*d*sizeof(int)));
    

    int elements = newNq/(d*d*d);       // Number of points that a box can contain
    printf("Elements: %d\n",elements);

    start = cpuSecond();
    for (int i=0; i<Nq; i++){
        int coX_C,coY_C,coZ_C;
        coX_C = corpus[i]/side;
        coY_C = corpus[i+Nq]/side;
        coZ_C = corpus[i+2*Nq]/side;
        int boxC = coX_C + coY_C*d + coZ_C*d*d;     // Find the corresponding box of the point
        if ((boxC >= d*d*d) || (boxC < 0)){
            printf("Error, non existing box\n");
            exit(1);
        }
        boxesC[boxC*elements + auxC[boxC]] = corpus[i];     // --> Store the point inside the box 
        boxesC[boxC*elements + auxC[boxC] + newNq] = corpus[i + Nq];
        boxesC[boxC*elements + auxC[boxC] + 2*newNq] = corpus[i + 2*Nq];  // <--
        auxC[boxC]++;
        if (auxC[boxC]>=elements){
            printf("We have problem\n");
            exit(1);
        }    
    }
    elaps = cpuSecond() - start;
    printf("Time elapsed for hashing the total number of points on host: %f seconds\n",elaps);

    // for (int i=0; i<newNq; i++){
    //     printf("BoxesC[%d]: (%f,%f,%f)\n",i, boxesC[i], boxesC[i+newNq], boxesC[i+2*newNq]);
    // }

    // for (int i=0; i<d*d*d; i++){
    //     printf("AuxC: %d\n",auxC[i]);
    // }

    float* distances;
    CHECK(hipMallocManaged(&distances,Nq*sizeof(float)));

    start = cpuSecond();
    findNN<<<((Nq+1023)/1024),1024>>>(queries,Nq,boxesC,newNq,elements,side,auxC,d,distances);
    // findNN<<<1,1>>>(queries,Nq,boxesC,newNq,elements,side,auxC,d);
    CHECK(hipDeviceSynchronize());
    elaps = cpuSecond() - start;
    printf("Time elapsed for knn on device: %f seconds\n",elaps);

    int* flag;
    CHECK(hipMallocManaged(&flag,sizeof(int)));

    start = cpuSecond();
    validate<<<((Nq+1023)/1024),1024>>>(queries,corpus,Nq,flag,distances);
    CHECK(hipDeviceSynchronize());
    elaps = cpuSecond() - start;
    printf("Time elapsed for brute force knn on device: %f seconds\n",elaps);

    if (*flag != 1){
        printf("Validation PASSED\n");
    }
    else{
        printf("Validation FAILED\n");
    }

    if (Nq < (1<<16)){
        start = cpuSecond();
        for (int j=0;j<Nq;j++){
            float min = 10000;
            for (int i=0;i<Nq;i++){
                float sum = 0;
                for (int k=0; k<3; k++){
                    sum += powf(queries[j+k*Nq]-corpus[i+k*Nq],2);
                }
                float distance = powf(sum,0.5);
                if (distance < min){
                    min = distance;
                }
            }
            // printf("Minimum distance for query[%d]: %f\n",j,min);
            if (fabsf(min-distances[j]) > 0.000001){
                printf("Validation failed\n");
                exit(1);
            }
        }
        elaps = cpuSecond() - start;
        printf("Time elapsed for brute force knn on host: %f seconds\n",elaps);
        printf("Validation PASSED\n");
    }
   


    hipFree(boxesC);
    hipFree(auxC);
    hipFree(queries);
    hipFree(corpus);
    hipFree(distances);
    hiprandDestroyGenerator(gen);
    return 0;
}
